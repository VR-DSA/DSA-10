#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>
#include <arpa/inet.h>

#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"

#include "fitsio.h"

#define NINTS 100
#define NCALCS 100
const double diffobs = 3000.0;

float summed_vis[27500];

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out, multilog_t * log);

int docrab(double *crabs, double samp);
int docal(double *cals, double samp);

int docrab(double *crabs, double samp) {

  for (int i=0;i<NCALCS;i++)
    if ((crabs[i]-samp<diffobs) && (crabs[i]-samp>-diffobs)) return 1;

  return 0;

}

int docal(double *cals, double samp) {

  for (int i=0;i<NCALCS;i++)
    if ((cals[i]-samp<diffobs) && (cals[i]-samp>-diffobs)) return 1;

  return 0;

}

void usage()
{
  fprintf (stdout,
	   "dsaX_image [options]\n"
	   " -c core   bind process to CPU core\n"
	   " -f filename [default test.fits]\n"
	   " -o freq of chan 1 [default 1530.0]\n"
	   " -h        print usage\n");
}

void dsaX_dbgpu_cleanup (dada_hdu_t * in, multilog_t * log) {

  if (dada_hdu_unlock_read (in) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in\n");
    }
  dada_hdu_destroy (in);

}

int main (int argc, char *argv[]) {

  /* DADA defs */
  dada_hdu_t* hdu_in = 0;
  multilog_t* log = 0;
  key_t in_key = 0x0000eada;

  // load in mjds
  FILE *fin;
  double *cals;
  crabs = (double *)malloc(sizeof(double)*NCALCS);
  cals = (double *)malloc(sizeof(double)*NCALCS);
  fin=fopen("/mnt/nfs/runtime/crab_mjds.dat","r");
  for (int i=0;i<NCALCS;i++) fscanf(fin,"%lf\n",&crabs[i]);
  fclose(fin);
  fin=fopen("/mnt/nfs/runtime/cal_mjds.dat","r");
  for (int i=0;i<NCALCS;i++) fscanf(fin,"%lf\n",&cals[i]);
  fclose(fin);
  cout << "Read Crab and cal MJDs" << endl;
  
  // command line
  int arg = 0;
  int core = -1;
  int nsamps = 384;
  float fch1 = 1530.0;
  int npts = 55*125*2*2;
  int nsamps_gulp = 384;
  char fnam[300], foutnam[400];
  sprintf(fnam,"/mnt/nfs/data/alltest");
  
  while ((arg=getopt(argc,argv,"c:f:o:h")) != -1)
    {
      switch (arg)
	{
	case 'c':
	  if (optarg)
	    {
	      core = atoi(optarg);
	      break;
	    }
	  else
	    {
	      printf ("ERROR: -c flag requires argument\n");
	      return EXIT_FAILURE;
	    }
	case 'f':
	  strcpy(fnam,optarg);
	  break;
	case 'o':
	  fch1 = atof(optarg);
	  break;
	case 'h':
	  usage();
	  return EXIT_SUCCESS;
	}
    }

  // DADA stuff
  
  log = multilog_open ("dsaX_image", 0);
  
  multilog_add (log, stderr);

  multilog (log, LOG_INFO, "dsaX_image: creating hdu\n");

  hdu_in  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0) {
    printf ("dsaX_image: could not connect to dada buffer\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in) < 0) {
    printf ("dsaX_image: could not lock to dada buffer\n");
    return EXIT_FAILURE;
  }

  hdu_out  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out, out_key);
  if (dada_hdu_connect (hdu_out) < 0) {
    printf ("dsaX_ftus: could not connect to output  buffer\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out) < 0) {
    fprintf (stderr, "dsaX_ftus: could not lock to output buffer\n");
    return EXIT_FAILURE;
  }

  // Bind to cpu core
  if (core >= 0)
    {
      printf("binding to core %d\n", core);
      if (dada_bind_thread_to_core(core) < 0)
	printf("dsaX_image: failed to bind to core %d\n", core);
    }

  int observation_complete=0;

  // more DADA stuff - deal with headers
  
  uint64_t header_size = 0;

  // read the headers from the input HDUs and mark as cleared
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  if (!header_in)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  char * header_out = ipcbuf_get_next_write (hdu_out->header_block);
  if (!header_out)
    {
      multilog(log, LOG_ERR, "could not get next header block [output]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }
  memcpy (header_out, header_in, header_size);
  if (ipcbuf_mark_filled (hdu_out->header_block, header_size) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block filled [output]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  // set up
  FILE *ftime;
  int fctr = 0, integration = 0;
  char tstamp[100];
  double mjd, samp = 0.0;
  fitsfile *fptr;
  int rownum = 1;
  int fwrite = 0;
  int status=0;
  int crabtime = 0, caltime = 0;
  
  // data stuff
  uint64_t block_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in->data_block);
  uint64_t out_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_out->data_block);
  uint64_t bytes_read = 0, block_id, written;
  char *block;
  float  *out_block;
  out_block = (float *)malloc(sizeof(float)*out_size/4);
  float *data, m1r, m1i, m2r, m2i;

  // mean vis, and vis def
  float ****mvis, *****vis;
  mvis = (float ****)malloc(sizeof(float ***)*45);
  for (int i=0;i<45;i++) {
    mvis[i] = (float ***)malloc(sizeof(float **)*125);
    for (int j=0;j<125;j++) {
      mvis[i][j] = (float **)malloc(sizeof(float *)*2);
      for (int k=0;k<2;k++) {
	mvis[i][j][k] = (float *)malloc(sizeof(float)*2);
	mvis[i][j][k][0] = 0.;
	mvis[i][j][k][1] = 0.;
      }
    }
  }
  vis = (float *****)malloc(sizeof(float ****)*nsamps);
  for (int ti=0;ti<nsamps;ti++) {
    vis[ti] = (float ****)malloc(sizeof(float ***)*45);
    for (int i=0;i<45;i++) {
      vis[ti][i] = (float ***)malloc(sizeof(float **)*125);
      for (int j=0;j<125;j++) {
	vis[ti][i][j] = (float **)malloc(sizeof(float *)*2);
	for (int k=0;k<2;k++) {
	  vis[ti][i][j][k] = (float *)malloc(sizeof(float)*2);
	}
      }
    }
  }
  int mblock=10, bi;


  // set up lookup table for triplets

  // find indices of actual baselines
  int bases[45], a1[45], a2[45];
  int bct = 0, allct = 0;
  for (int i=0;i<10;i++) {
    for (int j=0;j<=i;j++) {
      if (i!=j) {
	bases[bct] = allct;
	a1[bct] = j;
	a2[bct] = i;
	bct++;
      }
      allct++;
    }
  }
  cout << "printing all baseline numbers ";
  for (int i=0;i<45;i++) cout << bases[i] << " " << a1[i] << " " << a2[i] << endl;
  cout << endl;
  bct = 0;

  // brute force all triplets
  int trips[120][3], tct = 0;
  for (int i=0;i<10;i++) {
    for (int j=0;j<i;j++) {
      for (int k=0;k<j;k++) {

	//triplet is from antennas i,j,k
	
	// loop over bases three times to find correct baselines

	for (int l=0;l<45;l++) {
	  if ((a1[l]==i && a2[l]==j) || (a1[l]==j && a2[l]==i))
	    trips[tct][0] = l;
	}

	for (int l=0;l<45;l++) {
	  if ((a1[l]==i && a2[l]==k) || (a1[l]==k && a2[l]==i))
	    trips[tct][1] = l;
	}

	for (int l=0;l<45;l++) {
	  if ((a1[l]==k && a2[l]==j) || (a1[l]==j && a2[l]==k))
	    trips[tct][2] = l;
	}

	tct++;
	
      }
    }
  }

  cout << "printing all triplet numbers ";
  for (int i=0;i<120;i++) cout << trips[i][0] << " " << trips[i][1] << " " << trips[i][2] << endl;
  cout << endl;
 
  
  // start things

  multilog(log, LOG_INFO, "dsaX_image: starting observation\n");

  while (!observation_complete) {

    block = ipcio_open_block_read (hdu_in->data_block, &bytes_read, &block_id);
    data = (float *)block; // order is [384 time, 55 baseline, 125 freq, 2 pol, 2 ri]

    // sum input visibilities
    for (int i=0;i<npts;i++) summed_vis[i] = 0.;
    for (int i=0;i<nsamps_gulp;i++) {
      for (int j=0;j<npts;j++)
	summed_vis[j] += data[i*npts+j];
    }

    // three file writing cases: initial (fctr=0), crab and cal (fctr > 0)

    // CASE FOR CRAB OR CAL FILE

    if (fctr > 0) {

      // check for source
      crabtime = docrab(crabs,samp);
      caltime = docal(cals,samp);
      if (crabtime || caltime) fwrite = 1;
      else {
	if (fwrite == 1) {
	  integration = 0;
	  cout << "Completed file " << fctr << endl;
	  fctr++;
	}
	fwrite = 0;
      }
      

      // do file writing
      if (fwrite) {

	if (crabtime) multilog(log, LOG_INFO, "dsaX_final: crabtime\n");
	if (caltime) multilog(log, LOG_INFO, "dsaX_final: caltime\n");

	// create file
	if (integration==0) {
	
	  if (crabtime) {
	    sprintf(foutnam,"%s_%s_crab_%d.fits",fnam,tstamp,fctr);
	  }
	  if (caltime) {
	    sprintf(foutnam,"%s_%s_cal_%d.fits",fnam,tstamp,fctr);
	  }
	  cout << "main: opening new file " << foutnam << endl;
	  rownum=1;
	
	  char *ttype[] = {"VIS"};
	  char *tform[] = {"27500E"}; // assumes classic npts
	  char *tunit[] = {"\0"};
	  char *antennas = "3-7-2-10-1-4-5-8-6-9";
	
	  
	  char extname[] = "DATA";
	  fits_create_file(&fptr, foutnam, &status);
	  if (status) cerr << "create_file FITS error " << status << endl;
	  fits_create_tbl(fptr, BINARY_TBL, 0, 1, ttype, tform, tunit, extname, &status);
	  if (status) cerr << "create_tbl FITS error " << status << endl;
	  fits_write_key(fptr, TDOUBLE, "MJD", &mjd, "Start MJD", &status);
	  float mytsamp = nsamps_gulp*8.192e-6*128.;
	  fits_write_key(fptr, TFLOAT, "TSAMP", &mytsamp, "Sample time (s)", &status);
	  fits_write_key(fptr, TFLOAT, "FCH1", &fch1, "Frequency (MHz)", &status);
	  fits_write_key(fptr, TSTRING, "Antennas", &antennas[0], "Antennas", &status);
	  
	  if (status) cerr << "FITS error " << status << endl;
      
	  fits_close_file(fptr, &status);

	}
      
	// write to file
   
	fits_open_table(&fptr, foutnam, READWRITE, &status);
	fits_write_col(fptr, TFLOAT, 1, rownum, 1, 66000, summed_vis, &status);
	if (status) cerr << "FITS error in write " << status << endl;
	rownum += 1;
	fits_update_key(fptr, TINT, "NAXIS2", &rownum, "", &status);
	fits_close_file(fptr, &status);
	integration++;

      }
      
    }

    // CASE FOR FIRST FILE
    else if (fctr==0) {

      // if first integration of first file
      if (samp == 0) {
	
	// get start time, and convert others to samples
	ftime=fopen("/mnt/nfs/runtime/UTC_START.txt","r");
	fscanf(ftime,"%lf\n",&mjd);
	fscanf(ftime,"%[^\n]",&tstamp[0]);
	fclose(ftime);

	for (int i=0;i<NCALCS;i++) {
	  crabs[i] = (crabs[i]-mjd)*86400./(nsamps_gulp*8.192e-6*128.);
	  cals[i] = (cals[i]-mjd)*86400./(nsamps_gulp*8.192e-6*128.);
	}

	// open file
	
	sprintf(foutnam,"%s_%s_%d.fits",fnam,tstamp,fctr);
	cout << "main: opening new file " << foutnam << endl;
	rownum=1;
	
	char *ttype[] = {"VIS"};
	char *tform[] = {"27500E"}; // assumes classic npts
	char *tunit[] = {"\0"};
	char *antennas = "3-7-2-10-1-4-5-8-6-9";
	  
	char extname[] = "DATA";
	fits_create_file(&fptr, foutnam, &status);
	if (status) cerr << "create_file FITS error " << status << endl;
	fits_create_tbl(fptr, BINARY_TBL, 0, 1, ttype, tform, tunit, extname, &status);
	if (status) cerr << "create_tbl FITS error " << status << endl;
	fits_write_key(fptr, TDOUBLE, "MJD", &mjd, "Start MJD", &status);
	float mytsamp = nsamps_gulp*8.192e-6*128.;
	fits_write_key(fptr, TFLOAT, "TSAMP", &mytsamp, "Sample time (s)", &status);
	fits_write_key(fptr, TFLOAT, "FCH1", &fch1, "Frequency (MHz)", &status);
	fits_write_key(fptr, TSTRING, "Antennas", &antennas[0], "Antennas", &status);

	if (status) cerr << "FITS error " << status << endl;
      
	fits_close_file(fptr, &status);

      }
      
      // write to file
   
      fits_open_table(&fptr, foutnam, READWRITE, &status);
      fits_write_col(fptr, TFLOAT, 1, rownum, 1, 66000, summed_vis, &status);
      if (status) cerr << "FITS error in write " << status << endl;
      rownum += 1;
      fits_update_key(fptr, TINT, "NAXIS2", &rownum, "", &status);
      fits_close_file(fptr, &status);
      integration++;

      if (integration==NINTS) {
	integration=0;
	cout << "Completed file " << fctr << endl;
	fctr++;
      }
      
    }

    // update mjd and samp
    mjd += nsamps_gulp*128.*8.192e-6/86400.;
    samp += 1.0;
    
    // find mean vis
    if (bct<mblock) {

      multilog(log, LOG_INFO, "dsaX_image: finding mean vis %d of %d\n",bct+1,mblock);
      
      for (int t_idx=0;t_idx<nsamps;t_idx++) {
	for (int b_idx=0;b_idx<45;b_idx++) {
	  bi = bases[b_idx];
	  for (int f_idx=0;f_idx<125;f_idx++) {
	    for (int p_idx=0;p_idx<2;p_idx++) {
	      for (int r_idx=0;r_idx<2;r_idx++) {

		mvis[b_idx][f_idx][p_idx][r_idx] += data[t_idx*27500+bi*500+f_idx*4+p_idx*2+r_idx];
		
	      }
	    }
	  }
	}
      }
      for (int b_idx=0;b_idx<45;b_idx++) {
	bi = bases[b_idx];
	for (int f_idx=0;f_idx<125;f_idx++) {
	  for (int p_idx=0;p_idx<2;p_idx++) {
	    for (int r_idx=0;r_idx<2;r_idx++) {
	      
	      mvis[b_idx][f_idx][p_idx][r_idx] /= 1.*nsamps*mblock;
	      
	    }
	  }
	}
      }

      bct++;
      
    }
    // do bispectrum
    else {
    
      // subtract mean vis
      for (int t_idx=0;t_idx<nsamps;t_idx++) {
	for (int b_idx=0;b_idx<45;b_idx++) {
	  bi = bases[b_idx];
	  for (int f_idx=0;f_idx<125;f_idx++) {
	    for (int p_idx=0;p_idx<2;p_idx++) {
	      for (int r_idx=0;r_idx<2;r_idx++) {

		vis[t_idx][b_idx][f_idx][p_idx][r_idx] = data[t_idx*27500+bi*500+f_idx*4+p_idx*2+r_idx]-mvis[b_idx][f_idx][p_idx][r_idx];
		
	      }
	    }
	  }
	}
      }
    
      // for each time bin, sum over triple product of each baseline triplet. 
      // indices for baselines to use are in trips

      // form bispectrum as two complex multiply steps.
      // m1_r = ar*br - ai*bi, m1_i = ar*bi + ai*br
      // m2_r = m1_r*cr + m1_i*ci, m2_i = m1_i*cr - m1_r*ci

      // main loop
      for (int t_idx=0;t_idx<nsamps;t_idx++) {
	
	for (int f_idx=0;f_idx<125;f_idx++) {

	  out_block[t_idx*125+f_idx] = 0.;

	  for (int trip_idx=0;trip_idx<120;trip_idx++) {
	    for (int p_idx=0;p_idx<2;p_idx++) {

	      m1r = vis[t_idx][trips[trip_idx][0]][f_idx][p_idx][0] * vis[t_idx][trips[trip_idx][1]][f_idx][p_idx][0] - vis[t_idx][trips[trip_idx][0]][f_idx][p_idx][1] * vis[t_idx][trips[trip_idx][1]][f_idx][p_idx][1];
	      m1i = vis[t_idx][trips[trip_idx][0]][f_idx][p_idx][0] * vis[t_idx][trips[trip_idx][1]][f_idx][p_idx][1] + vis[t_idx][trips[trip_idx][0]][f_idx][p_idx][1] * vis[t_idx][trips[trip_idx][1]][f_idx][p_idx][0];
	      m2r = m1r * vis[t_idx][trips[trip_idx][2]][f_idx][p_idx][0] + m1i * vis[t_idx][trips[trip_idx][2]][f_idx][p_idx][1];
	      m2i = -m1r * vis[t_idx][trips[trip_idx][2]][f_idx][p_idx][1] + m1i * vis[t_idx][trips[trip_idx][2]][f_idx][p_idx][0];
	      
	      out_block[t_idx*125+f_idx] += sqrt(m2r*m2r+m2i*m2i);

	    }
	  }
	}
	
      }

      // write out_block
      written = ipcio_write (hdu_out->data_block, (char *)(out_block), out_size);
      if (written < out_size)
	{
	  multilog(log, LOG_INFO, "main: failed to write all data to datablock [output]\n");
	  dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
	  return EXIT_FAILURE;
	}
      
      }
        
    // close off loop
    if (bytes_read < block_size)
      observation_complete = 1;

    ipcio_close_block_read (hdu_in->data_block, bytes_read);
    
  }

  free(mvis);
  free(vis);
  free(crabs);
  free(cals);
  dsaX_dbgpu_cleanup(hdu_in, hdu_out, log);
 
}
