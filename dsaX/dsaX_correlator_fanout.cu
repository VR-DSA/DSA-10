#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>

#include "sock.h"
#include "tmutil.h"
#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"

#include "fitsio.h"

#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

void transpose(int *dst, const int *src, size_t n, size_t p);

void transpose(int *dst, const int *src, size_t n, size_t p)  {
    size_t block = 32;
    for (size_t i = 0; i < n; i += block) {
        for(size_t j = 0; j < p; ++j) {
            for(size_t b = 0; b < block && i + b < n; ++b) {
                dst[j*n + i + b] = src[(i + b)*p + j];
            }
        }
    }
}

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out1, dada_hdu_t * out2, dada_hdu_t * out3, dada_hdu_t * out4, dada_hdu_t * out5, dada_hdu_t * out6, multilog_t * log);
int dada_bind_thread_to_core (int core);

int dada_bind_thread_to_core(int core)
{

  cpu_set_t set;
  pid_t tpid;

  CPU_ZERO(&set);
    CPU_SET(core, &set);
      tpid = syscall(SYS_gettid);

  if (sched_setaffinity(tpid, sizeof(cpu_set_t), &set) < 0) {
    printf("failed to set cpu affinity: %s", strerror(errno));
    return -1;
  }
  
  CPU_ZERO(&set);
  if ( sched_getaffinity(tpid, sizeof(cpu_set_t), &set) < 0 ) {
    printf("failed to get cpu affinity: %s", strerror(errno));
    return -1;
  }

  return 0;
}

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out1, dada_hdu_t * out2, dada_hdu_t * out3, dada_hdu_t * out4, dada_hdu_t * out5, multilog_t * log)
{
  
  if (dada_hdu_unlock_read (in) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in\n");
    }
  dada_hdu_destroy (in);

  if (dada_hdu_unlock_write (out1) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out1\n");
    }
  dada_hdu_destroy (out1);

  if (dada_hdu_unlock_write (out2) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out2\n");
    }
  dada_hdu_destroy (out2);

  if (dada_hdu_unlock_write (out3) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out3\n");
    }
  dada_hdu_destroy (out3);

  if (dada_hdu_unlock_write (out4) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out4\n");
    }
  dada_hdu_destroy (out4);

  if (dada_hdu_unlock_write (out5) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out5\n");
    }
  dada_hdu_destroy (out5);
  
}

void usage()
{
  fprintf (stdout,
	   "dsaX_beamformer_fanout [options]\n"
	   " -c core   bind process to CPU core\n"
	   " -n OUT_NSAMPS [default 98304]\n"
	   " -h print usage\n");
}

int main (int argc, char *argv[]) {
  
  /* DADA Header plus Data Unit */
  dada_hdu_t* hdu_in = 0;
  dada_hdu_t* hdu_out1 = 0;
  dada_hdu_t* hdu_out2 = 0;
  dada_hdu_t* hdu_out3 = 0;
  dada_hdu_t* hdu_out4 = 0;
  dada_hdu_t* hdu_out5 = 0;
  
  /* DADA Logger */
  multilog_t* log = 0;
  
  // data block HDU keys
  key_t in_key = 0x0000caca;
  key_t out_key1 = 0x0000dbda;
  key_t out_key2 = 0x0000dcda;
  key_t out_key3 = 0x0000ddda;
  key_t out_key4 = 0x0000ebda;
  key_t out_key5 = 0x0000ecda;

  // command line arguments
  int core = -1;
  int arg=0;
  int OUT_NSAMPS = 98304;
  
  while ((arg=getopt(argc,argv,"c:n:h")) != -1)
    {
      switch (arg)
	{
	case 'c':
	  core = atoi(optarg);
	  break;
	case 'n':
	  OUT_NSAMPS = atoi(optarg);
	  break;
	case 'h':
	  usage();
	  return EXIT_SUCCESS;
	}
    }

  // DADA stuff
  
  log = multilog_open ("dsaX_correlator_fanout", 0);
  multilog_add (log, stderr);

  // open connection to the in/read DBs
  
  hdu_in  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0) {
    printf ("dsaX_correlator_fanout: could not connect to input buffer\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in) < 0) {
    printf ("dsaX_correlator_fanout: could not lock to input buffer\n");
    return EXIT_FAILURE;
  }

  hdu_out1  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out1, out_key1);
  if (dada_hdu_connect (hdu_out1) < 0) {
    printf ("dsaX_correlator_fanout: could not connect to output buffer1\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out1) < 0) {
    dsaX_dbgpu_cleanup (hdu_in, hdu_out1, hdu_out2, hdu_out3, hdu_out4, hdu_out5, log);
    fprintf (stderr, "dsaX_correlatir_fanout: could not lock to output buffer1\n");
    return EXIT_FAILURE;
  }

  hdu_out2  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out2, out_key2);
  if (dada_hdu_connect (hdu_out2) < 0) {
    printf ("dsaX_correlator_fanout: could not connect to output buffer2\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out2) < 0) {
    dsaX_dbgpu_cleanup (hdu_in, hdu_out1, hdu_out2, hdu_out3, hdu_out4, hdu_out5, log);
    fprintf (stderr, "dsaX_correlator_fanout: could not lock to output buffer2\n");
    return EXIT_FAILURE;
  }

  hdu_out3  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out3, out_key3);
  if (dada_hdu_connect (hdu_out3) < 0) {
    printf ("dsaX_correlator_fanout: could not connect to output buffer3\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out3) < 0) {
    dsaX_dbgpu_cleanup (hdu_in, hdu_out1, hdu_out2, hdu_out3, hdu_out4, hdu_out5, log);
    fprintf (stderr, "dsaX_correlator_fanout: could not lock to output buffer3\n");
    return EXIT_FAILURE;
  }

  hdu_out4  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out4, out_key4);
  if (dada_hdu_connect (hdu_out4) < 0) {
    printf ("dsaX_correlator_fanout: could not connect to output buffer4\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out4) < 0) {
    dsaX_dbgpu_cleanup (hdu_in, hdu_out1, hdu_out2, hdu_out3, hdu_out4, hdu_out5, log);
    fprintf (stderr, "dsaX_correlator_fanout: could not lock to output buffer4\n");
    return EXIT_FAILURE;
  }

  hdu_out5  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out5, out_key5);
  if (dada_hdu_connect (hdu_out5) < 0) {
    printf ("dsaX_correlator_fanout: could not connect to output buffer5\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out5) < 0) {
    dsaX_dbgpu_cleanup (hdu_in, hdu_out1, hdu_out2, hdu_out3, hdu_out4, hdu_out5, log);
    fprintf (stderr, "dsaX_correlator_fanout: could not lock to output buffer5\n");
    return EXIT_FAILURE;
  }
  
  // Bind to cpu core
  if (core >= 0)
    {
      printf("binding to core %d\n", core);
      if (dada_bind_thread_to_core(core) < 0)
	printf("dsaX_correlator_fanout: failed to bind to core %d\n", core);
    }

  // more DADA stuff - deal with headers
  
  uint64_t header_size = 0;

  // read the header from the input HDU, and get output header blocks
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  char * header_out1 = ipcbuf_get_next_write (hdu_out1->header_block);
  char * header_out2 = ipcbuf_get_next_write (hdu_out2->header_block);
  char * header_out3 = ipcbuf_get_next_write (hdu_out3->header_block);
  char * header_out4 = ipcbuf_get_next_write (hdu_out4->header_block);
  char * header_out5 = ipcbuf_get_next_write (hdu_out5->header_block);
 
  // copy the in header to the out header
  memcpy (header_out1, header_in, header_size);
  memcpy (header_out2, header_in, header_size);
  memcpy (header_out3, header_in, header_size);
  memcpy (header_out4, header_in, header_size);
  memcpy (header_out5, header_in, header_size);

  // mark the header buffers as cleared/filled
  ipcbuf_mark_cleared (hdu_in->header_block);
  ipcbuf_mark_filled (hdu_out1->header_block, header_size);
  ipcbuf_mark_filled (hdu_out2->header_block, header_size);
  ipcbuf_mark_filled (hdu_out3->header_block, header_size);
  ipcbuf_mark_filled (hdu_out4->header_block, header_size);
  ipcbuf_mark_filled (hdu_out5->header_block, header_size);
  
  // main reading loop

  multilog(log, LOG_INFO, "main: starting read\n");
  bool observation_complete=0;
  bool started_recording=0;
  int nints=0;
  
  // variables
  char * h_data;
  int *h_outdata, *h_intdata, *iter;
  h_outdata = (int *)malloc(sizeof(int)*OUT_NSAMPS*300);
  iter = (int *)malloc(sizeof(int)*OUT_NSAMPS*2048);
  uint64_t block_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in->data_block);
  uint64_t  bytes_read = 0;
  uint64_t block_id;
  int bytes_to_write = OUT_NSAMPS*250*2*2;
  uint64_t written=0;

  while (!observation_complete) {

    // read a DADA block
    h_data = ipcio_open_block_read (hdu_in->data_block, &bytes_read, &block_id);
    h_intdata = (int *)h_data;
    transpose(iter,h_intdata,OUT_NSAMPS,2048);
    
    // do the writing
    thrust::copy(iter+350*OUT_NSAMPS,iter+600*OUT_NSAMPS,h_outdata);
    written = ipcio_write (hdu_out1->data_block, (char *) h_outdata, bytes_to_write);
    thrust::copy(iter+600*OUT_NSAMPS,iter+850*OUT_NSAMPS,h_outdata);
    written = ipcio_write (hdu_out2->data_block, (char *) h_outdata, bytes_to_write);
    thrust::copy(iter+850*OUT_NSAMPS,iter+1100*OUT_NSAMPS,h_outdata);
    written = ipcio_write (hdu_out3->data_block, (char *) h_outdata, bytes_to_write);
    thrust::copy(iter+1100*OUT_NSAMPS,iter+1350*OUT_NSAMPS,h_outdata);
    written = ipcio_write (hdu_out4->data_block, (char *) h_outdata, bytes_to_write);
    thrust::copy(iter+1350*OUT_NSAMPS,iter+1600*OUT_NSAMPS,h_outdata);
    written = ipcio_write (hdu_out5->data_block, (char *) h_outdata, bytes_to_write);

    // close block for reading
    ipcio_close_block_read (hdu_in->data_block, bytes_read);
    multilog(log, LOG_INFO, "main: finished a block\n");

    if (bytes_read < block_size) {
      observation_complete = 1;
      multilog(log, LOG_INFO, "main: finished, with bytes_read %llu < expected %llu\n", bytes_read, block_size);
      break;
    }

  }

  dsaX_dbgpu_cleanup (hdu_in, hdu_out1, hdu_out2, hdu_out3, hdu_out4, hdu_out5, log);
  free(h_outdata);
  free(iter);
  
}

