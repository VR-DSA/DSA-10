#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>
#include <omp.h>

#include "dada_cuda.h"
#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"


#include "cube/cube.h"
#include "xgpu.h"

#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

// if NBLOCKS*NTHREADS is changed, the kernel may need to be changed
#define NBLOCKS 100
#define NTHREADS 32

void dsaX_dbgpu_cleanup (dada_hdu_t * in1, dada_hdu_t * in2, dada_hdu_t * in3, dada_hdu_t * in4, dada_hdu_t * in5, dada_hdu_t * out, multilog_t * log);
int dada_bind_thread_to_core (int core);
void simple_extract(Complex *mat, float *output);

// n is number in each input. typically nsamps_gulp*300
// inputs i1-i5 are SNAPs, in [frequency, time, (ant, pol, ri)] order
// output needs to be [time, frequency, ant, pol, ri]
__global__
void massage(char *i1, char *output, int nsamps_gulp) {

  int n = nsamps_gulp*250*8*5;
  int idx = blockIdx.x*blockDim.x + threadIdx.x; // global index
  int m = n/NBLOCKS/NTHREADS; // number of ints to process per thread

  int sidx, sfq_idx, st_idx, in1, oidx, ti, sn_idx, t1;
  
  for (int i=0;i<m;i++) {

    sn_idx = (int)((idx*m+i) / (250*nsamps_gulp*8)); // snap idx
    t1 = (int)((idx*m+i) % (250*nsamps_gulp*8)); // remainder from multi snaps
    sfq_idx = (int)(t1/8/nsamps_gulp); // fq_idx
    st_idx = (int)((t1/8)  % nsamps_gulp); // t_idx
    sidx = (int)(t1 % 8); // char idx
    oidx = 64*(st_idx*250+sfq_idx)+sidx+8*sn_idx; // output idx at start
    ti = (idx*m+i);

    output[oidx] = i1[ti]/16;
   
    
  }
    

}



int dada_bind_thread_to_core(int core)
{

  cpu_set_t set;
  pid_t tpid;

  CPU_ZERO(&set);
    CPU_SET(core, &set);
      tpid = syscall(SYS_gettid);

  if (sched_setaffinity(tpid, sizeof(cpu_set_t), &set) < 0) {
      printf("failed to set cpu affinity: %s", strerror(errno));
          return -1;
	    }

  CPU_ZERO(&set);
    if ( sched_getaffinity(tpid, sizeof(cpu_set_t), &set) < 0 ) {
        printf("failed to get cpu affinity: %s", strerror(errno));
	    return -1;
	      }

  return 0;
}



void usage()
{
  fprintf (stdout,
	   "dsaX_correlator [options]\n"
	   " -c core   bind process to CPU core\n"
	   " -h        print usage\n");
}

int main (int argc, char *argv[]) {
  
  /* DADA Header plus Data Unit */
  dada_hdu_t* hdu_in1 = 0;
  dada_hdu_t* hdu_in2 = 0;
  dada_hdu_t* hdu_in3 = 0;
  dada_hdu_t* hdu_in4 = 0;
  dada_hdu_t* hdu_in5 = 0;
  dada_hdu_t* hdu_out = 0;

  /* DADA Logger */
  multilog_t* log = 0;

  int core = -1;

  // input data block HDU key
  key_t in_key1 = 0x0000bdad;
  key_t in_key2 = 0x0000bdcd;
  key_t in_key3 = 0x0000bddd;
  key_t in_key4 = 0x0000bbbb;
  key_t in_key5 = 0x0000bbab;
  key_t out_key = 0x0000eada;

  int arg = 0;
  int nsamps_gulp=49152;
  int nout=202752000;

  while ((arg=getopt(argc,argv,"c:n:h")) != -1)
    {
      switch (arg)
	{
	case 'c':
	  if (optarg)
	    {
	      core = atoi(optarg);
	      break;
	    }
	  else
	    {
	      printf ("ERROR: -c flag requires argument\n");
	      return EXIT_FAILURE;
	    }
	case 'n':
	  nout = atoi(optarg);
	  break;
	case 'h':
	  usage();
	  return EXIT_SUCCESS;
	}
    }

  
  
  // DADA stuff
  
  log = multilog_open ("dsaX_massager", 0);

  multilog_add (log, stderr);

  multilog (log, LOG_INFO, "dsaX_massager: creating in hdus\n");

  // open connection to the in/read DBs
  
  hdu_in1  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in1, in_key1);
  if (dada_hdu_connect (hdu_in1) < 0) {
    printf ("dsaX_massager: could not connect to dada buffer1\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in1) < 0) {
    printf ("dsaX_massager: could not lock to dada buffer1\n");
    return EXIT_FAILURE;
  }

  hdu_in2  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in2, in_key2);
  if (dada_hdu_connect (hdu_in2) < 0) {
    printf ("dsaX_massager: could not connect to dada buffer2\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in2) < 0) {
    printf ("dsaX_massager: could not lock to dada buffer2\n");
    return EXIT_FAILURE;
  }

  hdu_in3  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in3, in_key3);
  if (dada_hdu_connect (hdu_in3) < 0) {
    printf ("dsaX_massager: could not connect to dada buffer3\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in3) < 0) {
    printf ("dsaX_massager: could not lock to dada buffer3\n");
    return EXIT_FAILURE;
  }

  hdu_in4  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in4, in_key4);
  if (dada_hdu_connect (hdu_in4) < 0) {
    printf ("dsaX_massager: could not connect to dada buffer4\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in4) < 0) {
    printf ("dsaX_massager: could not lock to dada buffer4\n");
    return EXIT_FAILURE;
  }

  hdu_in5  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in5, in_key5);
  if (dada_hdu_connect (hdu_in5) < 0) {
    printf ("dsaX_massager: could not connect to dada buffer5\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in5) < 0) {
    printf ("dsaX_massager: could not lock to dada buffer5\n");
    return EXIT_FAILURE;
  }

  hdu_out  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out, out_key);
  if (dada_hdu_connect (hdu_out) < 0) {
    printf ("dsaX_massager: could not connect to output  buffer\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out) < 0) {
    fprintf (stderr, "dsaX_massager: could not lock to output buffer\n");
    return EXIT_FAILURE;
  }
  
  
  // Bind to cpu core
  if (core >= 0)
    {
      printf("binding to core %d\n", core);
      if (dada_bind_thread_to_core(core) < 0)
	printf("dsaX_correlator: failed to bind to core %d\n", core);
    }

  bool observation_complete=0;
	
  // more DADA stuff - deal with headers
  
  uint64_t header_size = 0;

  // read the headers from the input HDUs and mark as cleared
  char * header_in1 = ipcbuf_get_next_read (hdu_in1->header_block, &header_size);
  if (!header_in1)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in1->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  char * header_in2 = ipcbuf_get_next_read (hdu_in2->header_block, &header_size);
  if (!header_in2)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in2->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  char * header_in3 = ipcbuf_get_next_read (hdu_in3->header_block, &header_size);
  if (!header_in3)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in3->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  char * header_in4 = ipcbuf_get_next_read (hdu_in4->header_block, &header_size);
  if (!header_in4)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in4->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  char * header_in5 = ipcbuf_get_next_read (hdu_in5->header_block, &header_size);
  if (!header_in5)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in5->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }

  // deal with output header
  char * header_out = ipcbuf_get_next_write (hdu_out->header_block);
  if (!header_out)
    {
      multilog(log, LOG_ERR, "could not get next header block [output]\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  memcpy (header_out, header_in1, header_size);
  if (ipcbuf_mark_filled (hdu_out->header_block, header_size) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block filled [output]\n");
      dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
      return EXIT_FAILURE;
    }
  
  
  uint64_t block_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in1->data_block);
  uint64_t  bytes_read1 = 0, bytes_read2 = 0, bytes_read3 = 0, bytes_read4 = 0, bytes_read5 = 0, written;
  uint64_t block_out = (uint64_t)(nout);
  uint64_t sgulp, ogulp;
  
  // set up

  // set up xgpu
  XGPUInfo xgpu_info;
  int syncOp = SYNCOP_DUMP;
  int xgpu_error = 0;
  xgpuInfo(&xgpu_info);
  XGPUContext context;
  context.array_h = NULL;
  context.matrix_h = NULL;
  xgpu_error = xgpuInit(&context, 0);
  if(xgpu_error) {
    multilog(log, LOG_ERR, "dsaX_xgpu: xGPU error %d\n", xgpu_error);
    dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
    return EXIT_FAILURE;
  }
  ComplexInput *array_h = context.array_h; // this is pinned memory
  Complex *cuda_matrix_h = context.matrix_h;
  float *output_vis = (float *)malloc(sizeof(float)*2*2*125*55);
  memset((char *)array_h,0,2*context.array_len);
  char *output_buffer = (char *)malloc(sizeof(char)*block_out);

  // host vectors for new data
  thrust::device_vector<char> d_i1(250*nsamps_gulp*8*5);
  thrust::device_vector<char> d_massage(nsamps_gulp * 250 * 16 * 2 * 2);
  uint64_t bytes_in_block = nsamps_gulp * 250 * 16 * 2 * 2;
  thrust::fill(d_massage.begin(),d_massage.end(),0);
  char *omassage = thrust::raw_pointer_cast(d_massage.data());
  uint64_t block_id1, block_id2, block_id3, block_id4, block_id5;
  char *block1, *block2, *block3, *block4, *block5;
  char *i1;
  i1 = thrust::raw_pointer_cast(d_i1.data());
  int nspec =250;

  // register stuff with gpu
  dada_cuda_dbregister(hdu_in1);
  dada_cuda_dbregister(hdu_in2);
  dada_cuda_dbregister(hdu_in3);
  dada_cuda_dbregister(hdu_in4);
  dada_cuda_dbregister(hdu_in5);
  
  // start everything
  
  multilog(log, LOG_INFO, "dsaX_massager: starting observation\n");

  while (!observation_complete) {

    block1 = ipcio_open_block_read (hdu_in1->data_block, &bytes_read1, &block_id1);
    block2 = ipcio_open_block_read (hdu_in2->data_block, &bytes_read2, &block_id2);
    block3 = ipcio_open_block_read (hdu_in3->data_block, &bytes_read3, &block_id3);
    block4 = ipcio_open_block_read (hdu_in4->data_block, &bytes_read4, &block_id4);
    block5 = ipcio_open_block_read (hdu_in5->data_block, &bytes_read5, &block_id5);
      
    // copy blocks to device
    thrust::copy(block1,block1+nspec*nsamps_gulp*8,d_i1.begin());
    thrust::copy(block2,block2+nspec*nsamps_gulp*8,d_i1.begin()+nspec*nsamps_gulp*8);
    thrust::copy(block3,block3+nspec*nsamps_gulp*8,d_i1.begin()+2*nspec*nsamps_gulp*8);
    thrust::copy(block4,block4+nspec*nsamps_gulp*8,d_i1.begin()+3*nspec*nsamps_gulp*8);
    thrust::copy(block5,block5+nspec*nsamps_gulp*8,d_i1.begin()+4*nspec*nsamps_gulp*8);

   
    // massage
    massage<<<NBLOCKS, NTHREADS>>>(i1,omassage,nsamps_gulp);
    hipDeviceSynchronize();
    
    // loop over sub-samples of input
    sgulp = 0;
    ogulp = 0;
    while (sgulp < bytes_in_block) {

      // select sub-gulp
      thrust::copy(d_massage.begin()+sgulp,d_massage.begin()+sgulp+128*250*64,(char *)array_h);
      
      // run xGPU
      xgpu_error = xgpuCudaXengine(&context, syncOp);
      if(xgpu_error) {
	multilog(log, LOG_ERR, "dsaX_xgpu: xGPU error %d\n", xgpu_error);
	dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
	return EXIT_FAILURE;
      }
      
      // reorder and extract output matrix
      simple_extract(cuda_matrix_h,output_vis);
      
      // copy to output buffer
      //memcpy(output_buffer+ogulp,(char *)output_vis,2*2*250*55*4);
      //memcpy(output_buffer+ogulp,(char *)cuda_matrix_h,136*250*4*2*4);
      // write to output
      written = ipcio_write (hdu_out->data_block, (char *)output_vis, 2*2*125*55*4);
      if (written < 2*2*125*55*4)
	{
	  multilog(log, LOG_INFO, "main: failed to write all data to datablock [output]\n");
	  dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);
	  return EXIT_FAILURE;
	}
      xgpuClearDeviceIntegrationBuffer(&context);

      // increment
      sgulp += 128*250*64;
      //ogulp += 2*2*250*55*4;
      //ogulp += 136*250*4*2*4;
       
    }

    multilog(log, LOG_INFO, "main: written block\n");
    
    
    if (bytes_read1 < block_size || bytes_read2 < block_size || bytes_read3 < block_size || bytes_read4 < block_size || bytes_read5 < block_size)
      observation_complete = 1;
    ipcio_close_block_read (hdu_in1->data_block, bytes_read1);
    ipcio_close_block_read (hdu_in2->data_block, bytes_read2);
    ipcio_close_block_read (hdu_in3->data_block, bytes_read3);
    ipcio_close_block_read (hdu_in4->data_block, bytes_read4);
    ipcio_close_block_read (hdu_in5->data_block, bytes_read5);

  }

  // unregister and free
  dada_cuda_dbunregister(hdu_in1);
  dada_cuda_dbunregister(hdu_in2);
  dada_cuda_dbunregister(hdu_in3);
  dada_cuda_dbunregister(hdu_in4);
  dada_cuda_dbunregister(hdu_in5);
  dsaX_dbgpu_cleanup (hdu_in1, hdu_in2, hdu_in3, hdu_in4, hdu_in5, hdu_out, log);

}


// assumes TRIANGULAR_ORDER for mat (f, baseline, pol, ri)
void simple_extract(Complex *mat, float *output) {

  int in_idx, out_idx;
  for (int bctr=0;bctr<55;bctr++) {
    for (int pol1=0;pol1<2;pol1++) {

      for (int f=0;f<125;f++) {

	out_idx = 2*((bctr*125+f)*2+pol1);
	in_idx = (2*f*136+bctr)*4+pol1*3;
	output[out_idx] = 0.5*(mat[in_idx].real + mat[in_idx+544].real);
	output[out_idx+1] = 0.5*(mat[in_idx].imag + mat[in_idx+544].imag);
	
      }
    }
  }

}


void dsaX_dbgpu_cleanup (dada_hdu_t * in1, dada_hdu_t * in2, dada_hdu_t * in3, dada_hdu_t * in4, dada_hdu_t * in5, dada_hdu_t * out, multilog_t * log)
{
  
  if (dada_hdu_unlock_read (in1) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in1\n");
    }
  dada_hdu_destroy (in1);

  if (dada_hdu_unlock_read (in2) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in2\n");
    }
  dada_hdu_destroy (in2);
  if (dada_hdu_unlock_read (in3) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in3\n");
    }
  dada_hdu_destroy (in3);
  if (dada_hdu_unlock_read (in4) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in4\n");
    }
  dada_hdu_destroy (in4);
  if (dada_hdu_unlock_read (in5) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in5\n");
    }
  dada_hdu_destroy (in5);

  if (dada_hdu_unlock_write (out) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out\n");
    }
  dada_hdu_destroy (out);
  
}
