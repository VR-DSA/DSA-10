#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>
#include <omp.h>

#include "dada_cuda.h"
#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"


#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out, multilog_t * log);
int dada_bind_thread_to_core (int core);
void massage (char * in, char * out, uint64_t block_size);

void massage (char * in, char * out, uint64_t block_size) {

  for (uint64_t i=0;i<block_size;i++) {

    out[2*i] = (char)(((unsigned char)(in[i]) & (unsigned char)(15)) << 4);
    out[2*i+1] = (char)((unsigned char)(in[i]) & (unsigned char)(240));

  }

}



int main (int argc, char *argv[]) {
  
  /* DADA Header plus Data Unit */
  dada_hdu_t* hdu_in = 0;
  dada_hdu_t* hdu_out = 0;

  /* DADA Logger */
  multilog_t* log = 0;

  int core = -1;

  // input data block HDU key
  key_t in_key = 0x0000eada;
  key_t out_key = 0x0000fada;
  int arg = 0;
  
  while ((arg=getopt(argc,argv,"c:k:o:h")) != -1)
    {
      switch (arg)
	{
	case 'c':
	  if (optarg)
	    {
	      core = atoi(optarg);
	      break;
	    }
	  else
	    {
	      printf ("ERROR: -c flag requires argument\n");
	      return EXIT_FAILURE;
	    }
	case 'k':
	  sscanf(optarg, "%x", &in_key);
	  break;
	case 'o':
	  sscanf(optarg, "%x", &out_key);
	  break;
	case 'h':
	  return EXIT_SUCCESS;
	}
    }
  
  // DADA stuff
  
  log = multilog_open ("dsaX_single", 0);

  multilog_add (log, stderr);

  multilog (log, LOG_INFO, "dsaX_single: creating hdus\n");

  hdu_in  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0) {
    printf ("dsaX: could not connect to dada buffer in\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in) < 0) {
    printf ("dsaX: could not lock to dada buffer inb\n");
    return EXIT_FAILURE;
  }

  hdu_out  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out, out_key);
  if (dada_hdu_connect (hdu_out) < 0) {
    printf ("dsaX: could not connect to output  buffer\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out) < 0) {
    fprintf (stderr, "dsaX: could not lock to output buffer\n");
    return EXIT_FAILURE;
  }

  // Bind to cpu core
  if (core >= 0)
    {
      printf("binding to core %d\n", core);
      if (dada_bind_thread_to_core(core) < 0)
	printf("dsaX: failed to bind to core %d\n", core);
    }

  bool observation_complete=0;

  uint64_t header_size = 0;

  // deal with headers
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  if (!header_in)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }
  if (ipcbuf_mark_cleared (hdu_in->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  char * header_out = ipcbuf_get_next_write (hdu_out->header_block);
  if (!header_out)
    {
      multilog(log, LOG_ERR, "could not get next header block [output]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }
  memcpy (header_out, header_in, header_size);
  if (ipcbuf_mark_filled (hdu_out->header_block, header_size) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block filled [output]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }
  
  uint64_t block_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in->data_block);
  uint64_t block_out = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_out->data_block);
  multilog(log, LOG_INFO, "main: have input and output block sizes %d %d\n",block_size,block_out);
  uint64_t  bytes_read = 0;
  
  char * block, * output_buffer;
  output_buffer = (char *)malloc(sizeof(char)*block_out);
  uint64_t written, block_id;

  // set up
  
  multilog(log, LOG_INFO, "dsaX_single: starting observation\n");

  while (!observation_complete) {

    // open block
    block = ipcio_open_block_read (hdu_in->data_block, &bytes_read, &block_id);

    // do the massage
    massage(block, output_buffer, block_size);
    
    // write to output
    written = ipcio_write (hdu_out->data_block, output_buffer, block_out);
    if (written < block_out)
      {
	multilog(log, LOG_INFO, "main: failed to write all data to datablock [output]\n");
	dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
	return EXIT_FAILURE;
      }

    //multilog(log, LOG_INFO, "main: written block\n");

    if (bytes_read < block_size)
      observation_complete = 1;

    ipcio_close_block_read (hdu_in->data_block, bytes_read);

  }


  free(output_buffer);
  dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
  
}


void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out, multilog_t * log)
{

  if (dada_hdu_unlock_read (in) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in\n");
    }
  dada_hdu_destroy (in);

  if (dada_hdu_unlock_write (out) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out\n");
    }
  dada_hdu_destroy (out);

}
