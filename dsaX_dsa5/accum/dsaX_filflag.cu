#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>

#include "dada_cuda.h"
#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"

#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "fitsio.h"
#include <src/sigproc.h>
#include <src/header.h>

#define NCHAN 2048

// binning for single-point flagging
#define BF 16
#define BT 25

FILE *output;

void send_string(char *string) /* includefile */
{
  int len;
  len=strlen(string);
  fwrite(&len, sizeof(int), 1, output);
  fwrite(string, sizeof(char), len, output);
}

void send_float(char *name,float floating_point) /* includefile */
{
  send_string(name);
  fwrite(&floating_point,sizeof(float),1,output);
}

void send_double (char *name, double double_precision) /* includefile */
{
  send_string(name);
  fwrite(&double_precision,sizeof(double),1,output);
}

void send_int(char *name, int integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(int),1,output);
}

void send_char(char *name, char integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(char),1,output);
}


void send_long(char *name, long integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(long),1,output);
}

void send_coords(double raj, double dej, double az, double za) /*includefile*/
{
  if ((raj != 0.0) || (raj != -1.0)) send_double("src_raj",raj);
  if ((dej != 0.0) || (dej != -1.0)) send_double("src_dej",dej);
  if ((az != 0.0)  || (az != -1.0))  send_double("az_start",az);
  if ((za != 0.0)  || (za != -1.0))  send_double("za_start",za);
}

void dsaX_dbgpu_cleanup (dada_hdu_t * hdu_in, dada_hdu_t * hdu_out, multilog_t * log);
int dada_bind_thread_to_core (int core);

// functor to do the scaling
__device__ float *s1, *s2, *s3;

struct da_functor
{

  __device__
  int operator()(const int x, const int y) const {

    int i = (int)(y % (NCHAN)); 

    if (i>1888) return __float2int_rn(64.0);
    else
      return __float2int_rn(x*s1[i]/s2[i]+s3[i]);
    
  }
};
int dada_bind_thread_to_core(int core)
{

  cpu_set_t set;
  pid_t tpid;

  CPU_ZERO(&set);
    CPU_SET(core, &set);
      tpid = syscall(SYS_gettid);

  if (sched_setaffinity(tpid, sizeof(cpu_set_t), &set) < 0) {
      fprintf(stderr, "failed to set cpu affinity: %s", strerror(errno));
          return -1;
	    }

  CPU_ZERO(&set);
    if ( sched_getaffinity(tpid, sizeof(cpu_set_t), &set) < 0 ) {
        fprintf(stderr, "failed to get cpu affinity: %s", strerror(errno));
	    return -1;
	      }

  return 0;
}

void usage()
{
  fprintf (stdout,
	   "dsaX_filflag [options]\n"
	   " -c core   bind process to CPU core\n"
	   " -f val    flagging level (0-nothing, 1-bandpassing, 2-birdies, 3-time-series, 4-brights)\n"
	   " -w        write filterbank file\n"
	   " -n name   file name base [default slog]\n"
	   " -k dada_in\n"
	   " -l dada_out\n"
	   " -h        print usage\n");
}

int main (int argc, char *argv[]) {

  hipSetDevice(0);
  
  /* DADA Header plus Data Unit */
  dada_hdu_t* hdu_in = 0;
  dada_hdu_t* hdu_out = 0;

  /* DADA Logger */
  multilog_t* log = 0;

  int core = -1;

  // input data block HDU key
  key_t in_key = 0x0000dada;

  // output data block HDU key
  key_t out_key = 0x0000eada;

  // command line
  int arg = 0;
  char fnam[200];
  sprintf(fnam,"slog");
  int filty=0;
  int flaglev=0;
  
  while ((arg=getopt(argc,argv,"c:k:l:f:n:wh")) != -1)
    {
      switch (arg)
	{
	case 'c':
	  if (optarg)
	    {
	      core = atoi(optarg);
	      break;
	    }
	  else
	    {
	      fprintf (stderr, "ERROR: -c flag requires argument\n");
	      return EXIT_FAILURE;
	    }
	case 'k':
	  if (sscanf (optarg, "%x", &in_key) != 1) {
	    fprintf (stderr, "dada_db: could not parse key from %s\n", optarg);
	    return EXIT_FAILURE;
	  }
	  break;
	case 'l':
	  if (sscanf (optarg, "%x", &out_key) != 1) {
	    fprintf (stderr, "dada_db: could not parse key from %s\n", optarg);
	    return EXIT_FAILURE;
	  }
	  break;
	case 'f':
	  flaglev=atoi(optarg);
	  if (flaglev<0 || flaglev>5) {
	    fprintf (stderr, "bad flaglev %s\n", optarg);
	    return EXIT_FAILURE;
	  }
	  break;
	case 'w':
	  filty=1;
	  break;
	case 'n':
	  strcpy(fnam,optarg);
	  break;
	case 'h':
	  usage();
	  return EXIT_SUCCESS;
	}
    }

  // DADA stuff

  log = multilog_open ("dsaX_filflag", 0);
  multilog_add (log, stderr);
  
  multilog (log, LOG_INFO, "dsaX_filflag: creating in hdu\n");
  // open connection to the in/read DB
  hdu_in  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0) {
    fprintf (stderr, "dsaX_spectrometer_reorder: could not connect to dada buffer\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in) < 0) {
    fprintf (stderr, "dsaX_spectrometer_reorder: could not lock to dada buffer\n");
    return EXIT_FAILURE;
  }

  // open connection to the out/write DB
  hdu_out = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out, out_key);
  if (dada_hdu_connect (hdu_out) < 0)
    {
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      fprintf (stderr, "dsaX_spectrometer_reorder: could not connect to eada buffer\n");
      return EXIT_FAILURE;
    }
  if (dada_hdu_lock_write(hdu_out) < 0)
    {
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      fprintf (stderr, "dsaX_spectrometer_reorder: could not lock4 to eada buffer\n");
      return EXIT_FAILURE;
    }

  if (core >= 0)
    {
      fprintf(stderr, "binding to core %d\n", core);
      if (dada_bind_thread_to_core(core) < 0)
	fprintf(stderr, "dsaX_spectrometer_reorder: failed to bind to core %d\n", core);
    }

  bool observation_complete=0;

  // more DADA stuff
  
  uint64_t header_size = 0;

  // read the header from the input HDU
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  if (!header_in)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  // now write the output DADA header
  char * header_out = ipcbuf_get_next_write (hdu_out->header_block);
  if (!header_out)
    {
      multilog(log, LOG_ERR, "could not get next header block [output]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  
  // copy the in header to the out header
  memcpy (header_out, header_in, header_size);

  // mark the input header as cleared
  if (ipcbuf_mark_cleared (hdu_in->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared [input]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  // mark the output header buffer as filled
  if (ipcbuf_mark_filled (hdu_out->header_block, header_size) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block filled [output]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }
  
  
  // setup vars
  uint64_t BLOCKSIZE = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in->data_block);
  // ASSUME THIS IS FOR BOTH INPUT AND OUTPUT
  uint64_t NSAMPS_GULP = BLOCKSIZE/NCHAN/2;
  uint64_t bytes_to_read;
  uint64_t block_id;
  char *   block;
  int bytes_to_write = BLOCKSIZE;
  uint64_t written=0;
  int ibyte, idx, idxo, idxs, idxg;
  
  // allocate memory to output
  unsigned short * out_data, * outout_data;
  thrust::host_vector<int> h_gulpdata(2048*NSAMPS_GULP);
  thrust::device_vector<int> d_gulpdata(2048*NSAMPS_GULP);
  thrust::device_vector<int> d_idx(2048*NSAMPS_GULP);
  thrust::sequence(d_idx.begin(),d_idx.end());
  out_data = (unsigned short *)malloc(sizeof(unsigned short)*BLOCKSIZE/2);
  outout_data = (unsigned short *)malloc(sizeof(unsigned short)*BLOCKSIZE/2);
  char *h_indata;
  uint64_t  bytes_read = 0, clipped, clipped_ts;
  int bp[2048], old_bp[2048], ts[NSAMPS_GULP];
  int ts_sum, ds_sum;
  // 11.8 is stddev of chi square with mean 64, 1 dof
  int thresh_ts = (int)((2048*64*BT)+(3.5*sqrt(2048.*BT*11.8)));
  float thresh_diff = (0.5/sqrt(NSAMPS_GULP*1.));
  float thresh_rms = 3.0;
  unsigned short repval = (unsigned short)(64);
  int clipthresh = (int)(64.*BT*BF+6.*sqrt(11.8*BT*BF));
  int clipthresh_ss = (int)(64.+6.*sqrt(11.8));
  unsigned short tmp;
  int intct = 0;
  int started_recording = 0;
  char cmd[200];
  uint64_t specnum = 0;
  int nints=0;
  float rmss[2048], tmpval;

  // to scale data
  thrust::host_vector<float> mult(NCHAN), sc(NCHAN), mlt(NCHAN);
  thrust::host_vector<float> mmult(NCHAN), msc(NCHAN), mmlt(NCHAN);
  thrust::device_vector<float> d_mult(NCHAN), d_sc(NCHAN), d_mlt(NCHAN);
  float *s_mult, *s_mlt, *s_sc;
  s_mult = thrust::raw_pointer_cast(&d_mult[0]);
  s_mlt = thrust::raw_pointer_cast(&d_mlt[0]);
  s_sc = thrust::raw_pointer_cast(&d_sc[0]);
  float bpscl = NSAMPS_GULP*64.;

  for (int i=0;i<NCHAN;i++) {
    sc[i] = 0.;
    mult[i] = bpscl;
    if (i<160) mult[i]=0.;
    if (i>515 && i<552) mult[i] = 0.;
    if (i>1208 && i<1232) mult[i] = 0.;
    if (i>1466 && i<1482) mult[i] = 0.;
    if (i>1777 && i<1794) mult[i] = 0.;
    if (i>1888) mult[i] = 0.;
    if (mult[i]==0.) mlt[i]=64.;
    else mlt[i] = 0.;    
  }

  // file for logging flagged spectra
  fitsfile *fptr;
  char fitsnam[100];
  int status=0;
  int rownum = 1;
  time_t rawtime;
  struct tm *info;
  time(&rawtime);
  info = localtime(&rawtime);
  double MJD = (double)(57754.+info->tm_yday+(info->tm_hour+8.)/24.+info->tm_min/(24.*60.)+info->tm_sec/(24.*60.*60.));
  sprintf(fitsnam,"/mnt/nfs/data/%s_%.3lf.fits",fnam,MJD);
  char *ttype[] = {"Spectra","Perc_ts","Perc_samp"};
  char *tform[] = {"2048E", "E", "E"}; 
  char *tunit[] = {"\0", "\0", "\0"};
  char extname[] = "spec_log";
  fits_create_file(&fptr, fitsnam, &status);
  if (status) cerr << "create_file FITS error " << status << endl;
  fits_create_tbl(fptr, BINARY_TBL, 0, 3, ttype, tform, tunit, extname, &status);
  if (status) cerr << "create_tbl FITS error " << status << endl;
  fits_write_key(fptr, TDOUBLE, "MJD", &MJD, "Start MJD", &status);
  float mytsamp = NSAMPS_GULP*1.31072e-4;
  fits_write_key(fptr, TFLOAT, "TSAMP", &mytsamp, "Sample time (s)", &status);

  if (status) cerr << "FITS error " << status << endl;
  else
    cout << "Opened FITS file " << fitsnam << endl;
  fits_close_file(fptr, &status);
  float out_bp[2048], out_pts[1], out_psamp[1];

  // output filterbank file if needed
  if (filty) {

    char filnam[300];
    sprintf(filnam,"/mnt/nfs/data/%s_%.3lf.fil",fnam,MJD);
    if (!(output = fopen(filnam,"wb"))) {
      printf("Couldn't open output file\n");
      return 0;
    }

    send_string("HEADER_START");
    send_string("source_name");
    send_string("TEST");
    send_int("machine_id",1);
    send_int("telescope_id",82);
    send_int("data_type",1); // filterbank data
    send_double("fch1",1530.0); // THIS IS CHANNEL 0 :)
    send_double("foff",-0.1220703125);
    send_int("nchans",2048);
    send_int("nbits",16);
    send_double("tstart",55000.0);
    send_double("tsamp",0.000131072);
    send_int("nifs",1);
    send_string("HEADER_END");
    
  }
  
  multilog(log, LOG_INFO, "main: starting observation\n");

  while (!observation_complete) {

    // open new file if needed.
    if (nints > 4395) {

      rownum = 1;
      nints=0;
      time(&rawtime);
      info = localtime(&rawtime);
      MJD = (double)(57754.+info->tm_yday+(info->tm_hour+8.)/24.+info->tm_min/(24.*60.)+info->tm_sec/(24.*60.*60.));
      sprintf(fitsnam,"/mnt/nfs/data/%s_%.3lf.fits",fnam,MJD);
      char *ttype[] = {"Spectra","Perc_ts","Perc_samp"};
      char *tform[] = {"2048E", "E", "E"}; 
      char *tunit[] = {"\0", "\0", "\0"};
      char extname[] = "spec_log";
      fits_create_file(&fptr, fitsnam, &status);
      if (status) cerr << "create_file FITS error " << status << endl;
      fits_create_tbl(fptr, BINARY_TBL, 0, 3, ttype, tform, tunit, extname, &status);
      if (status) cerr << "create_tbl FITS error " << status << endl;
      fits_write_key(fptr, TDOUBLE, "MJD", &MJD, "Start MJD", &status);
      fits_write_key(fptr, TFLOAT, "TSAMP", &mytsamp, "Sample time (s)", &status);

      if (status) cerr << "FITS error " << status << endl;
      else
	cout << "Opened FITS file " << fitsnam << endl;
      fits_close_file(fptr, &status);

    }

    // read a DADA block

    h_indata = ipcio_open_block_read (hdu_in->data_block, &bytes_read, &block_id);

    // FLAGLEV=0
    if (flaglev==0)
      memcpy(outout_data,h_indata,bytes_read);
    
    
    // deal with zero-ing stuff and setting up bandpasses.
    thrust::fill(h_gulpdata.begin(),h_gulpdata.end(),0);
    clipped = 0;
    clipped_ts = 0;
    for (int i=0;i<NSAMPS_GULP;i++) ts[i] = 0;
    for (int i=0;i<2048;i++) {
      if (started_recording) old_bp[i]=bp[i];
      else old_bp[i] = 0;
      bp[i] = 0;
    }

    // unpack data into h_gulpdata, and find current bp
    for (int k=0;k<NSAMPS_GULP;k++) {
		  
      for (int i=0;i<512;i++) {
	for (int j=0;j<4;j++) {
	  
	  idx = k*4096+i*8+j*2;
	  idxg = k*2048+i*4+j;
	  idxs = i*4+j;
	  tmp=0;
	  tmp |= (unsigned short)(h_indata[idx]) << 8;
	  tmp |= (unsigned short)(h_indata[idx+1]);
	  h_gulpdata[idxg] = (int)tmp;
	  bp[idxs] += (int)tmp;	     	    
	    
	}	
      }
	
    }

    // set up scaling by bandpass
    for (int i=0;i<2048;i++) {
      mmult[i] = mult[i];
      mmlt[i] = mlt[i];
      msc[i] = bp[i]*1.;
      // FLAGLEV=1
      if (old_bp[i]==0) {
	mmult[i] = 0.;
	mmlt[i] = 64.;
      }
      else if (((((float)(bp[i]-old_bp[i]))/((float)(old_bp[i]))>thresh_diff) || (((float)(old_bp[i]-bp[i]))/((float)(old_bp[i]))>thresh_diff)) && flaglev>1) {
	mmult[i] = 0.;
	mmlt[i] = 64.;
      }
    }
    thrust::copy(mmult.begin(),mmult.end(),d_mult.begin());
    thrust::copy(mmlt.begin(),mmlt.end(),d_mlt.begin());
    thrust::copy(msc.begin(),msc.end(),d_sc.begin());
    s_mult = thrust::raw_pointer_cast(&d_mult[0]);
    s_mlt = thrust::raw_pointer_cast(&d_mlt[0]);
    s_sc = thrust::raw_pointer_cast(&d_sc[0]);
    hipMemcpyToSymbol(HIP_SYMBOL(s1),&s_mult,sizeof(float *));
    hipMemcpyToSymbol(HIP_SYMBOL(s2),&s_sc,sizeof(float *));
    hipMemcpyToSymbol(HIP_SYMBOL(s3),&s_mlt,sizeof(float *));
    
    // do bandpass scaling of data
    thrust::copy(h_gulpdata.begin(),h_gulpdata.end(),d_gulpdata.begin());
    thrust::transform(d_gulpdata.begin(),d_gulpdata.end(),d_idx.begin(),d_gulpdata.begin(),da_functor());
    thrust::copy(d_gulpdata.begin(),d_gulpdata.end(),h_gulpdata.begin());

    // copy to out_data, and find ts
    for (int k=0;k<NSAMPS_GULP;k++) {
      for (int i=0;i<2048;i++) {
	idxo = k*2048+i;
	out_data[idxo] = (unsigned short)(h_gulpdata[idxo]);
	ts[k] += h_gulpdata[idxo];
      }
    }

    // FLAGLEV=1
    if (flaglev==1)
      memcpy(outout_data,out_data,bytes_read);

    // do variance flagging
    for (int k=0;k<NSAMPS_GULP;k++) {	 
      for (int i=0;i<2048;i++) {

	tmpval = static_cast<float>((out_data[k*2048+i]-64));
	rmss[i] += tmpval*tmpval;
	    
      }
	  
    }
    for (int i=0;i<2048;i++) {
      rmss[i] = sqrt(rmss[i]/(1.*NSAMPS_GULP));
      if (rmss[i] > 8.6*thresh_rms)  {
	for (int k=0;k<NSAMPS_GULP;k++) 
	  out_data[k*2048+i]=repval;
	rmss[i] = -1.;
      }
    }

    // FLAGLEV=2
    if (flaglev==2)
      memcpy(outout_data,out_data,bytes_read);
    
    // do ts flagging
    for (int k=0;k<NSAMPS_GULP/BT;k++) {

      // time-series flagging
      ts_sum = 0;
      for (int i=k*BT;i<(k+1)*BT;i++)
	ts_sum += ts[i];
      if (ts_sum>thresh_ts) {
	clipped_ts+=BT;
	for (int j=k*BT;j<(k+1)*BT;j++) {
	  for (int i=0;i<2048;i++) 
	    out_data[j*2048+i] = repval;
	}
      }
    }

    // FLAGLEV=3
    if (flaglev==3)
      memcpy(outout_data,out_data,bytes_read);

    // do single-point flagging
    for (int k=0;k<NSAMPS_GULP/BT;k++) {

      for (int i=0;i<2048/BF;i++) {
	ds_sum = 0;
	for (int j=k*BT;j<(k+1)*BT;j++) {
	  for (int l=i*BF;l<(i+1)*BF;l++)
	    ds_sum += out_data[j*2048+l];
	}
	if (ds_sum>clipthresh) {
	  for (int j=k*BT;j<(k+1)*BT;j++) {
	    for (int l=i*BF;l<(i+1)*BF;l++)
	      out_data[j*2048+l] = repval;
	    clipped+=BT*BF;
	  }
	}
      }
      
    }

    // FLAGLEV=4
    if (flaglev==4)
      memcpy(outout_data,out_data,bytes_read);

    // do logging
    fits_open_table(&fptr, fitsnam, READWRITE, &status);
    for (int i=0;i<2048;i++) {
      if (mmult[i]!=0) out_bp[i] = 1.*bp[i];
      else out_bp[i] = -1.*bp[i];
      if (rmss[i]==-1. && out_bp[i]>0.) out_bp[i] = -1.*bp[i];
    }
    out_pts[0] = (float)(100.*clipped_ts/(NSAMPS_GULP));
    out_psamp[0] = (float)(100.*clipped/(NSAMPS_GULP*2048));
    fits_write_col(fptr, TFLOAT, 1, rownum, 1, 2048, out_bp, &status);
    fits_write_col(fptr, TFLOAT, 2, rownum, 1, 1, out_pts, &status);
    fits_write_col(fptr, TFLOAT, 3, rownum, 1, 1, out_psamp, &status);
    if (status) cerr << "FITS error in write " << status << endl;

    rownum += 1;
    fits_update_key(fptr, TINT, "NAXIS2", &rownum, "", &status);
    fits_close_file(fptr, &status);
    nints++;      

    // do the start
    started_recording = 1;
        
    // DO THE WRITING TO BUFFER
    written = ipcio_write (hdu_out->data_block, (char *) outout_data, bytes_to_write);
    
    if (written < bytes_to_write)
      {
	multilog(log, LOG_INFO, "main: failed to write all data to datablock [output]\n");
	dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
	return EXIT_FAILURE;
      }

    // write to filterbank file
    if (filty)
      fwrite(outout_data,sizeof(unsigned short),NSAMPS_GULP*2048,output);

      
  }

  if (filty)
    fclose(output);
  
  dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
  free(out_data);
  free(outout_data);

}

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out, multilog_t * log)
{

  //dada_cuda_dbunregister (in);
  
  if (dada_hdu_unlock_read (in) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in\n");
    }
  dada_hdu_destroy (in);

  if (dada_hdu_unlock_write (out) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out\n");
    }
  dada_hdu_destroy (out);
}
